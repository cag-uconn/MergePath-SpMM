#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>


#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <fstream>
#include <functional>
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdint.h>
#include <string>
#include <vector>

#include "../../osdi-ae-graphs/cora.h"
using namespace std;

#define ROW_PTR 0
#define COL_IDX 1
#define WARP_SIZE 32

int DIM = 16;

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

__device__ inline 
void atomicAdd_F(float* address, float value)
{
  float old = value;  
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
}
struct CoordinateT {
    int x;
    int y;
};


CoordinateT MergePathSearch( int diagonal, volatile int* RP, int* NZ_INDICES, int num_rows, int nnz)
{
    
    int x_min = max(diagonal - nnz, 0);
    int x_max = min(diagonal, num_rows);

    while (x_min < x_max) {
        // so this is div by 2
        int pivot = (x_min + x_max) >> 1;
        if (RP[pivot] <= NZ_INDICES[diagonal - pivot - 1]) {
            x_min = pivot + 1;
        } 
        else {
            x_max = pivot;
        }
    }
    return CoordinateT{min(x_min, num_rows), diagonal - x_min};
}


std::vector<int *> generate_mp_sched(int num_threads) {

    int *feature_start_all = new int[num_threads];
    int *feature_end_all   = new int[num_threads];
    int *feature_start_num = new int[num_threads];
    int *feature_end_num   = new int[num_threads];
    
    int *start_row_all    = new int[num_threads];
    int *end_row_all       = new int[num_threads];
    int NZ_INDICES[FEATURE_TOTAL];

    for (int i = 0; i < num_threads; i++) {
        int core_id = i;

        int num_merge_items = NODE_ACT_NUM + FEATURE_TOTAL; 
        int items_per_thread = (num_merge_items + num_threads - 1) / num_threads;

        int diagonal = min(items_per_thread * core_id, num_merge_items);
        int diagonal_end = min(diagonal + items_per_thread, num_merge_items);
                                                                
        CoordinateT thread_coord = MergePathSearch(diagonal, feature_indices, NZ_INDICES, NODE_ACT_NUM, FEATURE_TOTAL);
        CoordinateT thread_coord_end = MergePathSearch(diagonal_end, feature_indices, NZ_INDICES, NODE_ACT_NUM, FEATURE_TOTAL);
    
        int start = thread_coord.x - 1;
        int end = thread_coord_end.x - 1;
        if (start < 0) start = 0;

        int num_features = 0;
        int features_start = 0;

        int feature_start = thread_coord.y;
        if (feature_indices[start] == feature_start) {
            feature_start = 0;
        }
        if (core_id == 0) {
            feature_start = 0;
        }

        int feature_end = thread_coord_end.y;
        if (feature_indices[end] == feature_end) {
            feature_end = 0;
        }

        if (feature_start != 0) {
            if (start == end && feature_end != 0) {
                num_features = feature_end - feature_start;
                feature_end = 0;
            }
            else {
                num_features = feature_indices[start + 1] - feature_start;
            }
            
        }
        int num_features_end = 0;
        if (feature_end != 0) num_features_end = feature_end - feature_indices[end];

        feature_start_all[core_id] = feature_start;
        feature_end_all[core_id]   = feature_end; 
        feature_start_num[core_id] = num_features;
        feature_end_num[core_id]   = num_features_end;   
        start_row_all[core_id]     = start;     
        end_row_all[core_id]       = end;       

    }
    cout << feature_start_all[0] << " " << feature_end_all[0] << " " << start_row_all[0]
    << " " << end_row_all[0]  << endl;
    return {feature_start_all, feature_end_all, 
            feature_start_num,
            feature_end_num,
            start_row_all, end_row_all};
    
}


__global__ void spmm_forward_cuda_kernel_mp(
    float *output,
    float *input, 
    int *row_pointers, 
    int *column_index, 
    int *degrees, 
    int *feature_start,
    int *feature_end,
    int *feature_start_num,
    int *feature_end_num,    
    int *start_row,
    int *end_row,
    int num_nodes, 
    int dim,
    int dimWorker,
    int warpPerBlock,
    int num_warps
) {

    int tid =  blockIdx.x * blockDim.x + threadIdx.x;  // global thread-id
    int warpId = tid / WARP_SIZE;                             // global warp-id
    //int block_warpId = threadIdx.x / WARP_SIZE;               // block warp-id
    int laneid = threadIdx.x % WARP_SIZE;                     // warp thread-id -- laneid

    // if (warpId == 0 && laneid == 0) {
    //     for (int i = 0; i < 100; i++)
    //     printf("%f\n", input[i * dim + laneid]);
    // } 
           
    if (warpId < num_warps) {
       
        int start = start_row[warpId];
        int end = end_row[warpId];
        int fstart = feature_start[warpId];
        int fstart_num = feature_start_num[warpId];
        int fend = feature_end[warpId];
        int fend_num = feature_end_num[warpId];

        float partial_results_start = 0;
        float  partial_results_end = 0;
        float output_temp = 0; 
        float degree_norm_inv = 0;
        float src_norm = 0;
        int index = 0;
        int num_features = 0;
        int features_start = 0;

        if (fstart != 0) {
            src_norm = 1;  
            
            for (int j = 0; j < fstart_num; j++) {
                index = column_index[fstart++];
                degree_norm_inv = __fmaf_rn(src_norm, 1, 0);
                partial_results_start += __fmaf_rn(degree_norm_inv, input[index * dim + laneid], 0); 
                            
            }
            if (warpId == 1 && laneid == 0) {
                
                printf("%d %d %d %d\n", fstart, fstart_num, fend, fend_num);
            }             
            atomicAdd_F((float*) &output[start * dim + laneid], partial_results_start);
            start = start + 1;
        }

        for (int i = start; i < end; i++) {
            src_norm = 1;
            output_temp = 0.0f;

            num_features = row_pointers[i + 1] - row_pointers[i];
            features_start = row_pointers[i]; 
            
            #pragma unroll
            for (int j = 0; j < num_features; j++) {
                index = column_index[features_start];
                degree_norm_inv = __fmaf_rn(src_norm, 1, 0);
                output_temp += __fmaf_rn(degree_norm_inv, input[index * dim + laneid], 0);

                features_start++;
            }

            output[i * dim + laneid] += output_temp;
        }             

        if (fend != 0) {
            src_norm = degrees[end];  

            #pragma unroll
            for (int j = 0; j < fend_num; j++) {
                index = column_index[fend++];
                degree_norm_inv = __fmaf_rn(src_norm, 1, 0);
                partial_results_end += __fmaf_rn(degree_norm_inv, input[index * dim + laneid], 0); 
            } 
     
            atomicAdd_F((float*) &output[end * dim + laneid], partial_results_end);
        }
        return;
    }
}


int main(int argc, char *argv[]) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); 

    if (argc < 2) {
        cout << "Please enter cost as well" << endl;
        exit(-1);
    }
    int cost = (atoi(argv[1]));
    int num_threads = (NODE_ACT_NUM + FEATURE_TOTAL) / cost;
    int num_nodes = NODE_NUM;

    /* Weight Matrix */
    float *h_input  = (float *) malloc(NODE_ACT_NUM * DIM * sizeof(float));
    float *h_output = (float *) malloc(NODE_ACT_NUM * DIM * sizeof(float)); 
    int *h_degrees = (int *) malloc(NODE_ACT_NUM * sizeof(int));

    for (int i = 0; i < NODE_ACT_NUM * DIM; i++) {
            h_input[i] = 1.0f;
    }
    for (int i = 0; i < NODE_ACT_NUM; i++) {
        h_degrees[i] = 2;
    }
    
   
    /* Device allocation */
    float *d_input, *d_output;
    int *d_row_pointer, *d_col_index, *d_degrees;
    int *d_feature_start, *d_feature_start_num, *d_feature_end, *d_feature_end_num;
    int *d_row_start, *d_row_end;
    auto mp_sched = generate_mp_sched(num_threads);

    hipMalloc((void**) &d_input, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemcpy(d_input, h_input, NODE_ACT_NUM * DIM * sizeof(float), hipMemcpyHostToDevice);
   
    hipMalloc((void**) &d_row_pointer, (NODE_ACT_NUM) * sizeof(int));
    hipMemcpy(d_row_pointer, feature_indices, (NODE_ACT_NUM) * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_col_index, (FEATURE_TOTAL) * sizeof(int));
    hipMemcpy(d_col_index, feature_indices_2, FEATURE_TOTAL * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_output, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemset(&d_output, 0, NODE_ACT_NUM * DIM * sizeof(float)); 

    hipMalloc((void**) &d_degrees, (NODE_ACT_NUM) * sizeof(int));
    hipMemcpy(d_degrees, h_degrees, NODE_ACT_NUM * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_feature_start, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemcpy(d_feature_start, mp_sched[0], num_threads * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_feature_end, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemcpy(d_feature_end, mp_sched[1], num_threads * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_feature_start_num, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemcpy(d_feature_start_num, mp_sched[2], num_threads * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_feature_end_num, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemcpy(d_feature_end_num, mp_sched[3], num_threads * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_row_start, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemcpy(d_row_start, mp_sched[4], num_threads * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_row_end, NODE_ACT_NUM * DIM * sizeof(float));
    hipMemcpy(d_row_end, mp_sched[5], num_threads * sizeof(int), hipMemcpyHostToDevice);

    /* Kernel Params */
    const int warpPerBlock = 8;
    const int block = warpPerBlock * WARP_SIZE; 
    const int grid = num_threads; 


    int repeats = 1;
    // for (int i = 0; i < num_threads; i++){
    //     cout << mp_sched[1][i] << endl;
    // }
    for (int i = 0; i < repeats; i++) {
        spmm_forward_cuda_kernel_mp<<<grid, block>>>(
            (float *) d_output, (float *) d_input, 
            (int *) d_row_pointer, (int *) d_col_index, (int *) d_degrees, 
            (int *) d_feature_start,
            (int *) d_feature_start_num,
            (int *) d_feature_end,
            (int *) d_feature_end_num,
            (int *) d_row_start,
            (int *) d_row_end,
            num_nodes, DIM, 32, 8, num_threads);
        hipDeviceSynchronize();
    }
    
    hipMemcpy(h_output, d_output, NODE_ACT_NUM * DIM * sizeof(float), hipMemcpyDeviceToHost);

    // for (int i = 0; i < NODE_NUM; i++) {
    //     for (int j = 0; j < DIM; j++) {
    //         std::cout << h_output[i * DIM + j] << ",";
    //     }
    //     std::cout << endl;
    // }
    return 0;
}