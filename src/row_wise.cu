#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>
#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <fstream>
#include <functional>
#include <iostream>
#include <math.h>
#include <sstream>
#include <stdint.h>
#include <string>
#include <vector>

using namespace std;

/* Total number of nodes */
int NODE_NUM = 0;
/* Total number of nodes in CSR */
int NODE_ACT_NUM = 0;
/* Total number of non-zeros */
int FEATURE_TOTAL = 0;

/* Row and column ptr */
int *row_ptr;
int *col_ptr;

const int WARP_SIZE = 32;
const int WARPS_PER_BLOCK = 8;
const int BLOCK = WARPS_PER_BLOCK * WARP_SIZE; 

__global__ void spmm_row_wise(
    float *output,
    float *input, 
    int *row_ptr, 
    int *col_ptr, 
    int *degrees, 
    int num_nodes, 
    int dimension,
    int dimWorker,
    int num_warps
);

void run_spmm_row_wise(
    float *output,
    float *input, 
    int *row_ptr, 
    int *col_ptr, 
    int *degrees, 
    int num_nodes, 
    int dimension,
    int dimWorker,
    int num_warps,
    int reapeats
);

int main(int argc, char *argv[]) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); 

    if (argc < 5) {
        cout << "Usage: ./application <input_file> <dimension> <num_warps> <repeat>" << endl;
        exit(-1);
    }

    ifstream matrix_file(argv[1]);
    int dimension = atoi(argv[2]);
    int num_warps = atoi(argv[3]);
    int repeats = atoi(argv[4]);


    /* Read the input file */
    string line;
    string cell;

    /* Count the toal number of nodes and non-zeros */
    getline(matrix_file, line); 
    stringstream lineStream(line);
    while(std::getline(lineStream,cell, ',')) {
        NODE_ACT_NUM++;
        FEATURE_TOTAL = stoi(cell);
    }
    NODE_NUM = NODE_ACT_NUM - 1;
    
    cout << "Total number of rows: " << NODE_NUM << " and non-zeros: " << FEATURE_TOTAL << endl;

    row_ptr   = (int *) malloc(NODE_ACT_NUM * sizeof(int));
    col_ptr = (int *) malloc(FEATURE_TOTAL * sizeof(int));
    
    /* Populate row and col ptrs*/
    matrix_file.seekg(ios_base::beg);
    {
        getline(matrix_file, line);
        int i = 0;
        stringstream lineStream(line);
        string cell;
    
        while(std::getline(lineStream,cell, ',')) {
            row_ptr[i] = stoi(cell);
            //cout << cell << endl;
            i++;
        }
        i = 0;
    }
    {
        getline(matrix_file, line);
        int i = 0;
        stringstream lineStream(line);
        string cell;
    
        while(std::getline(lineStream,cell, ',')) {
            col_ptr[i] = stoi(cell);
            //cout << cell << endl;
            i++;
        }
        i = 0;
    }
    /* This part of code remains the same for any kernel */
    /* Host side memory allocations */
    float *h_input    = (float *) malloc(NODE_ACT_NUM * dimension * sizeof(float));
    float *h_output   = (float *) malloc(NODE_ACT_NUM * dimension * sizeof(float)); 
    int   *h_degrees  = (int *) malloc(NODE_ACT_NUM * sizeof(int));
    
    /* Filling the input with dummy data */
    for (int i = 0; i < NODE_ACT_NUM * dimension; i++) {
        h_input[i]  = 1.0f;
        h_output[i] = 0.0f;
    }
    /* Calculating degree of each node */
    for (int i = 0; i < NODE_NUM; i++) {
        h_degrees[i] = row_ptr[i + 1] - row_ptr[i];
    }
   
    /* Device allocation */
    float *d_input, *d_output;
    int *d_row_ptr, *d_col_ptr, *d_degrees;
    
    hipMalloc((void**) &d_input, NODE_ACT_NUM * dimension * sizeof(float));
    hipMemcpy(d_input, h_input, NODE_ACT_NUM * dimension * sizeof(float), hipMemcpyHostToDevice);
    
    hipMalloc((void**) &d_row_ptr, (NODE_ACT_NUM) * sizeof(int));
    hipMemcpy(d_row_ptr, row_ptr, (NODE_ACT_NUM) * sizeof(int), hipMemcpyHostToDevice);
   
    hipMalloc((void**) &d_col_ptr, (FEATURE_TOTAL) * sizeof(int));
    hipMemcpy(d_col_ptr, col_ptr, FEATURE_TOTAL * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_output, NODE_ACT_NUM * dimension * sizeof(float));
    hipMemset(&d_output, 0, NODE_ACT_NUM * dimension * sizeof(float)); 
    
    hipMalloc((void**) &d_degrees, (NODE_ACT_NUM) * sizeof(int));
    hipMemcpy(d_degrees, h_degrees, NODE_ACT_NUM * sizeof(int), hipMemcpyHostToDevice);


    run_spmm_row_wise(d_output, d_input, d_row_ptr, d_col_ptr, d_degrees, NODE_NUM, 
    dimension, WARP_SIZE, num_warps, repeats);

    hipMemcpy(h_output, d_output, NODE_ACT_NUM * dimension * sizeof(float), hipMemcpyDeviceToHost);
    
    /* Verify the output */
    for (int i = 0; i < NODE_NUM; i++) {
        for (int j = 0; j < dimension; j++) {
            cout <<  (float)h_output[i * dimension + j] << "-";
        }
        std::cout << endl;
    }
    return 0;  
}

void run_spmm_row_wise(
    float *output,
    float *input, 
    int *row_ptr, 
    int *col_ptr, 
    int *degrees, 
    int num_nodes, 
    int dimension,
    int dimWorker,
    int num_warps,
    int repeats
) {
    int grid = num_warps;
    for (int i = 0; i < repeats; i++) {
            spmm_row_wise<<<grid, BLOCK>>>(
                (float *) output, (float *) input, 
                (int *) row_ptr, (int *) col_ptr, (int *) degrees, 
                num_nodes, dimension, dimWorker, grid);
            hipDeviceSynchronize();
    }
}

__global__ void spmm_row_wise(
    float *output,
    float *input, 
    int *row_ptr, 
    int *col_ptr, 
    int *degrees, 
    int num_nodes, 
    int dimension,
    int dimWorker,
    int num_warps
) {
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;  // global thread-id
    int warp_id = tid / WARP_SIZE;                      // global warp-id
    int lane_id = threadIdx.x % WARP_SIZE;              // warp thread-id -- laneid

    if (warp_id < num_warps) {
        /* Get the bounds */
        float step = (float)num_nodes / num_warps;
        int row_start = step * warp_id;
        int row_end = step * (warp_id + 1);
        
        int num_features = 0;
        int features_start = 0;
        float src_norm = 0;
        float degree_norm_inv = 0;
        int index = 0;
        float output_temp = 0;
        
        if (lane_id < dimension) {
            for (int i = row_start; i < row_end; i++) {
                output_temp = 0;
                num_features = row_ptr[i + 1] - row_ptr[i];
                features_start = row_ptr[i]; 
                src_norm = degrees[i];  
        
                #pragma unroll
                for (int j = 0; j < num_features; j++) {
                    index = col_ptr[features_start];
                    degree_norm_inv = __fmaf_rn(src_norm, degrees[index], 0);
                    output_temp += __fmaf_rn(degree_norm_inv, input[index * dimension + lane_id], 0);
                    features_start++;
                }
                output[i * dimension + lane_id] = output_temp;
            }    
        }
    }
}